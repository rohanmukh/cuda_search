//
// Created by rm38 on 3/2/20.
//
#include <hip/hip_runtime.h>
#define BLOCKSIZE 16

/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication
//
/////////////////////////////////////////////////////////////////////////////////////////

__global__
void MatVectMultiplication(const double *_device_Mat, const double *_device_Vect, int _matRowSize, int _vlength, double *_device_ResVect)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;


    if(tindex < _matRowSize)
    {
        int i;int m= tindex * _vlength;
        _device_ResVect[tindex]=0.00;
        for(i=0; i < _vlength; i++)
            _device_ResVect[tindex]+= _device_Mat[m + i] * _device_Vect[i];
    }

    __syncthreads();

}//end of MatVect device function

