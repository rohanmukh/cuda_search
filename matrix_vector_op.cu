//
// Created by rm38 on 3/2/20.
//
#include <hip/hip_runtime.h>
#include <cmath>
#define BLOCKSIZE 16

/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication
//
/////////////////////////////////////////////////////////////////////////////////////////

__global__
void MatVectMultiplication(const float *device_database_A, const float *device_database_B, const float *device_database_probY,
        const float *device_input_A, const float *device_input_B,
        int batch_size, int dimension, float *_device_ResVect)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;

    if(tindex < batch_size) {
        int m = tindex * dimension;
        _device_ResVect[tindex] = 0.00;
        for (int i = 0; i < dimension; i++) {
            _device_ResVect[tindex] += pow(device_input_B[i], 2) / (4 * device_input_A[0]); // additive ab1 1st item
            _device_ResVect[tindex] +=
                    pow(device_database_B[m + i], 2) / (4 * device_database_A[tindex]); // additive ab2 1st item
            _device_ResVect[tindex] -= pow(device_database_B[m + i] + device_input_B[i], 2) /
                                       (4 * (device_database_A[tindex] + device_input_A[0])); // subtractive ab_star 1st item
        }
        _device_ResVect[tindex] += 0.5 * dimension * log(-1 * (device_input_A[0]) / M_PI); // additive ab1 2nd item
        _device_ResVect[tindex] += 0.5 * dimension * log(-1 * (device_database_A[tindex]) / M_PI); // additive ab2_2nd item
        _device_ResVect[tindex] -= 0.5 * dimension * log(-1 * (device_database_A[tindex] + device_input_A[0]) /
                                                         M_PI); // subtractive ab_star 2nd item
        _device_ResVect[tindex] -= 0.5 * dimension * log(2 * M_PI); // subtractive cons
        _device_ResVect[tindex] += device_database_probY[tindex];
    }
    __syncthreads();

}//end of MatVect device function

