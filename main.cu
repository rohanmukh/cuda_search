#include "hip/hip_runtime.h"
/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     :  cuda-matrix-vector-multiplication.cu

  Objective   : Write CUDA program to compute Matrix-Vector multiplication.

  Input       : None

  Output      : Execution time in seconds , Gflops achieved

  Created     : August-2013

  E-mail      : hpcfte@cdac.in

****************************************************************************/


#include<cstdio>
#include<hip/hip_runtime.h>
#include "serial_code.h"
#include "utils.h"
#include "cuda_utils.h"

#define BLOCKSIZE 16
#define SIZE 1024

hipDeviceProp_t deviceProp;


double *device_Mat,*device_Vect,*device_ResVect;
int     vlength, matRowSize , matColSize;
int     size = SIZE;



/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication
//
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void MatVectMultiplication(double *_device_Mat, double *_device_Vect, int _matRowSize, int _vlength, double *_device_ResVect)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;


    if(tindex < _matRowSize)
    {
        int i;int m= tindex * _vlength;
        _device_ResVect[tindex]=0.00;
        for(i=0; i < _vlength; i++)
            _device_ResVect[tindex]+= _device_Mat[m + i] * _device_Vect[i];
    }

    __syncthreads();

}//end of MatVect device function



/*function to launch kernel*/
void launch_Kernel_MatVectMul()
{
/*          threads_per_block, blocks_per_grid  */
    int max=BLOCKSIZE*BLOCKSIZE;
    int BlocksPerGrid=matRowSize/max+1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(matRowSize%max==0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_Mat,device_Vect,matRowSize,vlength,device_ResVect);

}


/*main function*/
int main()
{
    // Vector length , Matrix Row and Col sizes..............
    vlength = matColSize = SIZE;
    matRowSize = SIZE;

    //  printf("this programs does computation of square matrix only\n");
    float elapsedTime;
    hipEvent_t start,stop;

    int device_Count=get_DeviceCount();
    printf("\n\nNumber of Devices : %d\n\n", device_Count);

    // Device Selection, Device 1: Tesla C1060
    hipSetDevice(0);

    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    printf("Using device %d: %s \n", device, deviceProp.name);



    /*allocating the memory for each matrix */
    double *host_Mat =new double[matRowSize*matColSize];
    double *host_Vect = new double[vlength];
    double *host_ResVect = new double[matRowSize];


    // ---------------checking host memory  for error..............................
    if(host_Mat==NULL)
        mem_error("host_Mat","vectmatmul",matRowSize*matColSize,"double");

    if(host_Vect==NULL)
        mem_error("host_Vect","vectmatmul",vlength,"double");

    if(host_ResVect==NULL)
        mem_error("host_ResVect","vectmatmul",matRowSize,"double");

    //--------------Initializing the input arrays..............
    fill_with_random_doubles(host_Mat, matRowSize * matColSize);
    fill_with_random_doubles(host_Vect, vlength);

    /* allocate memory for GPU events
       start = (hipEvent_t) malloc (sizeof(hipEvent_t));
       stop = (hipEvent_t) malloc (sizeof(hipEvent_t));
       if(start==NULL)
               mem_error("start","vectvectmul",1,"hipEvent_t");
       if(stop==NULL)
               mem_error("stop","vectvectmul",1,"hipEvent_t");*/

    //event creation...
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventCreate (&stop));

    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Mat, matRowSize*matColSize* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Vect, vlength* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_ResVect, matRowSize* sizeof(double)));

    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Mat, (void*)host_Mat, matRowSize*matColSize*sizeof(double) ,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Vect, (void*)host_Vect,vlength*sizeof(double),hipMemcpyHostToDevice));

    // Launching kernell..........
    CUDA_SAFE_CALL(hipEventRecord (start, 0));

    launch_Kernel_MatVectMul();

    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));


    // calling funtion for measuring Gflops & printing the result on screen
    float Tsec= 1.0e-3*elapsedTime;
    print_on_screen("MAT VECT MULTIPLICATION",Tsec,calculate_gflops(Tsec, size),size,1);


    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResVect,matRowSize*sizeof(double),hipMemcpyDeviceToHost));

    // CPU calculation..and checking error deviation....
    serial_code *cpu_user = new serial_code(matRowSize, matColSize, host_Mat, host_Vect, vlength, size);
    cpu_user->CPU_MatVect();
    relative_error(cpu_user->get_result(), host_ResVect, size);
    printf("\n ----------------------------------------------------------------------\n");

    /*free the memory from GPU */
    CUDA_SAFE_CALL(hipFree(device_Mat));
    CUDA_SAFE_CALL(hipFree(device_Vect));
    CUDA_SAFE_CALL(hipFree(device_ResVect));
    printf("mem freed\n");

    //free host memory----------
    free(host_Mat);
    free(host_Vect);
    free(host_ResVect);
    cpu_user->_free();

    return 0;
}// end of main

