/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     :  cuda-matrix-vector-multiplication.cu

  Objective   : Write CUDA program to compute Matrix-Vector multiplication.

  Input       : None

  Output      : Execution time in seconds , Gflops achieved

  Created     : August-2013

  E-mail      : hpcfte@cdac.in

****************************************************************************/


#include<cstdio>
#include<hip/hip_runtime.h>
#include "serial_code.h"
#include "utils.h"
#include "cuda_utils.h"
#include "gpu_ops.h"

#define SIZE 1024

hipDeviceProp_t deviceProp;

int vlength, matRowSize , matColSize;
int size = SIZE;

/*main function*/
int main()
{
    // Vector length , Matrix Row and Col sizes..............
    vlength = matColSize = SIZE;
    matRowSize = SIZE;


    /*allocating the memory for each matrix */
    double *host_Mat =new double[matRowSize*matColSize];
    double *host_Vect = new double[vlength];
    double *host_ResVect = new double[matRowSize];


    // ---------------checking host memory  for error..............................
    if(host_Mat==NULL)
        mem_error("host_Mat","vectmatmul",matRowSize*matColSize,"double");

    if(host_Vect==NULL)
        mem_error("host_Vect","vectmatmul",vlength,"double");

    if(host_ResVect==NULL)
        mem_error("host_ResVect","vectmatmul",matRowSize,"double");

    //--------------Initializing the input arrays..............
    fill_with_random_doubles(host_Mat, matRowSize * matColSize);
    fill_with_random_doubles(host_Vect, vlength);

    /* allocate memory for GPU events
       start = (hipEvent_t) malloc (sizeof(hipEvent_t));
       stop = (hipEvent_t) malloc (sizeof(hipEvent_t));
       if(start==NULL)
               mem_error("start","vectvectmul",1,"hipEvent_t");
       if(stop==NULL)
               mem_error("stop","vectvectmul",1,"hipEvent_t");*/


    // CUDA ops can start
    //  printf("this programs does computation of square matrix only\n");
    float elapsedTime;
    hipEvent_t stop;

    int device_Count=get_DeviceCount();
    printf("\n\nNumber of Devices : %d\n\n", device_Count);

    // Device Selection, Device 1: Tesla C1060
    hipSetDevice(0);

    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    printf("Using device %d: %s \n", device, deviceProp.name);



    //event creation...
    hipEvent_t start;
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventRecord (start, 0));
    CUDA_SAFE_CALL(hipEventCreate (&stop));

    gpu_ops *gpu_user = new gpu_ops(matRowSize, matColSize, host_Mat, host_Vect, vlength, size);
    gpu_user->allocate_memory();
    gpu_user->copy_to_device();
    gpu_user->launch_kernel(deviceProp);
    gpu_user->get_data_to_host(host_ResVect);

    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));


    // calling funtion for measuring Gflops & printing the result on screen
    float Tsec= 1.0e-3*elapsedTime;
    print_on_screen("MAT VECT MULTIPLICATION",Tsec,calculate_gflops(Tsec, size),size,1);


    // CPU calculation..and checking error deviation....
    serial_code *cpu_user = new serial_code(matRowSize, matColSize, host_Mat, host_Vect, vlength, size);
    cpu_user->CPU_MatVect();
    relative_error(cpu_user->get_result(), host_ResVect, size);
    printf("\n ----------------------------------------------------------------------\n");

    //free host memory----------
    free(host_Mat);
    free(host_Vect);
    free(host_ResVect);
    gpu_user->_free();
    cpu_user->_free();

    return 0;
}// end of main

