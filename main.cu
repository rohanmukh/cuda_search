#include "hip/hip_runtime.h"
/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     :  cuda-matrix-vector-multiplication.cu

  Objective   : Write CUDA program to compute Matrix-Vector multiplication.

  Input       : None

  Output      : Execution time in seconds , Gflops achieved

  Created     : August-2013

  E-mail      : hpcfte@cdac.in

****************************************************************************/


#include<cstdio>
#include<hip/hip_runtime.h>
#include "serial_code.h"
#include "mem_error.h"

#define BLOCKSIZE 16
#define SIZE 1024
#define EPS 1.0e-15

hipDeviceProp_t deviceProp;


double *host_Mat,*host_Vect,*host_ResVect;
double *device_Mat,*device_Vect,*device_ResVect;
int     vlength ,matRowSize , matColSize;
int     device_Count;
int     size = SIZE;



/*calculate Gflops*/
double calculate_gflops(float &Tsec)
{
    float gflops=(1.0e-9 * (( 2.0 * size*size )/Tsec));
    return gflops;
}



/*Check for safe return of all calls to the device */
void CUDA_SAFE_CALL(hipError_t call)
{
    hipError_t ret = call;
    //printf("RETURN FROM THE CUDA CALL:%d\t:",ret);
    switch(ret)
    {
        case hipSuccess:
            //              printf("Success\n");
            break;
            /*      case hipErrorInvalidValue:
                                    {
                                    printf("ERROR: InvalidValue:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }
                    case hipErrorInvalidDevicePointer:
                                    {
                                    printf("ERROR:Invalid Device pointeri:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }
                    case hipErrorInvalidMemcpyDirection:
                                    {
                                    printf("ERROR:Invalid memcpy direction:%i.\n",__LINE__);
                                    exit(-1);
                                    break;
                                    }                       */
        default:
        {
            printf(" ERROR at line :%i.%d' ' %s\n",__LINE__,ret,hipGetErrorString(ret));
            exit(-1);
            break;
        }
    }
}


/*free memory*/
void dfree(double * arr[],int len)
{
    for(int i=0;i<len;i++)
        CUDA_SAFE_CALL(hipFree(arr[i]));
    printf("mem freed\n");
}

/* function to calculate relative error*/
void relError(double* dRes,double* hRes,int size)
{
    double relativeError=0.0,errorNorm=0.0;
    int flag=0;
    int i;

    for( i = 0; i < size; ++i) {
        if (fabs(hRes[i]) > fabs(dRes[i]))
            relativeError = fabs((hRes[i] - dRes[i]) / hRes[i]);
        else
            relativeError = fabs((dRes[i] - hRes[i]) / dRes[i]);

        if (relativeError > EPS && relativeError != 0.0e+00 )
        {
            if(errorNorm < relativeError)
            {
                errorNorm = relativeError;
                flag=1;
            }
        }

    }
    if( flag == 1)
    {
        printf(" \n Results verfication : Failed");
        printf(" \n Considered machine precision : %e", EPS);
        printf(" \n Relative Error                  : %e\n", errorNorm);

    }
    else
        printf("\n Results verfication : Success\n");

}


/*prints the result in screen*/
void print_on_screen(char * program_name,float tsec,double gflops,int size,int flag)//flag=1 if gflops has been calculated else flag =0
{
    printf("\n---------------%s----------------\n",program_name);
    printf("\tSIZE\t TIME_SEC\t Gflops\n");
    if(flag==1)
        printf("\t%d\t%f\t%lf\t",size,tsec,gflops);
    else
        printf("\t%d\t%lf\t%lf\t",size,"---","---");

}

/*funtion to check blocks per grid and threads per block*/
void check_block_grid_dim(hipDeviceProp_t devProp,dim3 blockDim,dim3 gridDim)
{

    if( blockDim.x >= devProp.maxThreadsDim[0] || blockDim.y >= devProp.maxThreadsDim[1] || blockDim.z >= devProp.maxThreadsDim[2] )
    {
        printf("\nBlock Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2]);
        exit(-1);
    }

    if( gridDim.x >= devProp.maxGridSize[0] || gridDim.y >= devProp.maxGridSize[1] || gridDim.z >= devProp.maxGridSize[2] )
    {
        printf("\nGrid Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
        exit(-1);
    }
}


/*Get the number of GPU devices present on the host */
int get_DeviceCount()
{
    int count;
    hipGetDeviceCount(&count);
    return count;
}


/*Fill in the vector with double precision values */
void fill_dp_vector(double* vec,int size)
{
    int ind;
    for(ind=0;ind<size;ind++)
        vec[ind]=drand48();
}


/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication
//
/////////////////////////////////////////////////////////////////////////////////////////
__global__ void MatVectMultiplication(double *_device_Mat, double *_device_Vect, int _matRowSize, int _vlength, double *_device_ResVect)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;


    if(tindex < _matRowSize)
    {
        int i;int m= tindex * _vlength;
        _device_ResVect[tindex]=0.00;
        for(i=0; i < _vlength; i++)
            _device_ResVect[tindex]+= _device_Mat[m + i] * _device_Vect[i];
    }

    __syncthreads();

}//end of MatVect device function



/*function to launch kernel*/
void launch_Kernel_MatVectMul()
{
/*          threads_per_block, blocks_per_grid  */


    int max=BLOCKSIZE*BLOCKSIZE;
    int BlocksPerGrid=matRowSize/max+1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(matRowSize%max==0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_Mat,device_Vect,matRowSize,vlength,device_ResVect);

}


/*main function*/
int main()
{
    // Vector length , Matrix Row and Col sizes..............
    vlength = matColSize = SIZE;
    matRowSize = SIZE;

    //  printf("this programs does computation of square matrix only\n");
    float elapsedTime,Tsec;
    hipEvent_t start,stop;

    device_Count=get_DeviceCount();
    printf("\n\nNUmber of Devices : %d\n\n", device_Count);

    // Device Selection, Device 1: Tesla C1060
    hipSetDevice(0);

    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    printf("Using device %d: %s \n", device, deviceProp.name);



    /*allocating the memory for each matrix */
    host_Mat =new double[matRowSize*matColSize];
    host_Vect = new double[vlength];
    host_ResVect = new double[matRowSize];


    // ---------------checking host memory  for error..............................
    if(host_Mat==NULL)
        mem_error("host_Mat","vectmatmul",matRowSize*matColSize,"double");

    if(host_Vect==NULL)
        mem_error("host_Vect","vectmatmul",vlength,"double");

    if(host_ResVect==NULL)
        mem_error("host_ResVect","vectmatmul",matRowSize,"double");

    //--------------Initializing the input arrays..............
    fill_dp_vector(host_Mat,matRowSize*matColSize);
    fill_dp_vector(host_Vect,vlength);

    /* allocate memory for GPU events
       start = (hipEvent_t) malloc (sizeof(hipEvent_t));
       stop = (hipEvent_t) malloc (sizeof(hipEvent_t));
       if(start==NULL)
               mem_error("start","vectvectmul",1,"hipEvent_t");
       if(stop==NULL)
               mem_error("stop","vectvectmul",1,"hipEvent_t");*/

    //event creation...
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventCreate (&stop));

    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Mat, matRowSize*matColSize* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Vect, vlength* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_ResVect, matRowSize* sizeof(double)));

    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Mat, (void*)host_Mat, matRowSize*matColSize*sizeof(double) ,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Vect, (void*)host_Vect,vlength*sizeof(double),hipMemcpyHostToDevice));

    // Launching kernell..........
    CUDA_SAFE_CALL(hipEventRecord (start, 0));

    launch_Kernel_MatVectMul();

    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));

    Tsec= 1.0e-3*elapsedTime;

    // calling funtion for measuring Gflops
    calculate_gflops(Tsec);

    //printing the result on screen
    print_on_screen("MAT VECT MULTIPLICATION",Tsec,calculate_gflops(Tsec),size,1);


    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResVect,matRowSize*sizeof(double),hipMemcpyDeviceToHost));

    // CPU calculation..and checking error deviation....

    serial_code *cpu_user = new serial_code(matRowSize, matColSize, host_Mat, host_Vect, vlength, size);
    cpu_user->CPU_MatVect();
    relError(cpu_user->get_result(), host_ResVect, size);
    printf("\n ----------------------------------------------------------------------\n");

    /*free the memory from GPU */
    double *array[3];
    array[0]=device_Mat;
    array[1]=device_Vect;
    array[2]=device_ResVect;
    dfree(array,3);

    //free host memory----------
    free(host_Mat);
    free(host_Vect);
    free(host_ResVect);
    cpu_user->_free();

    return 0;
}// end of main

