/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     :  cuda-matrix-vector-multiplication.cu

  Objective   : Write CUDA program to compute Matrix-Vector multiplication.

  Input       : None

  Output      : Execution time in seconds , Gflops achieved

  Created     : August-2013

  E-mail      : hpcfte@cdac.in

****************************************************************************/


#include<cstdio>
#include<hip/hip_runtime.h>
#include "serial_code.h"
#include "utils.h"
#include "cuda_utils.h"
#include "gpu_manager.h"
#include "host_ops.h"

#define SIZE 1024


int vlength, matRowSize , matColSize;
int size = SIZE;

/*main function*/
int main()
{
    // Vector length , Matrix Row and Col sizes..............
    vlength = matColSize = SIZE;
    matRowSize = SIZE;

    host_ops *host_system = new host_ops(matRowSize, matColSize, vlength);
    host_system->fill_with_random_data();


    gpu_manager* manager = new gpu_manager(0, matRowSize, matColSize, vlength);
    manager->copy_data(host_system->host_Mat, host_system->host_Vect);
    float time_sec = manager->compute_and_store(host_system->host_ResVect);


    // calling funtion for measuring Gflops & printing the result on screen
    print_on_screen("MAT VECT MULTIPLICATION",time_sec, calculate_gflops(time_sec, size),size,1);


    // CPU calculation..and checking error deviation....
    serial_code *cpu_user = new serial_code(matRowSize, matColSize, host_system->host_Mat, host_system->host_Vect, vlength, size);
    cpu_user->CPU_MatVectMult();
    relative_error(cpu_user->get_result(), host_system->host_ResVect, size);
    printf("\n ----------------------------------------------------------------------\n");

    host_system->_free();
    manager->_free();
    cpu_user->_free();

    return 0;
}// end of main

