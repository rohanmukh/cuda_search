/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     :  cuda-matrix-vector-multiplication.cu

  Objective   : Write CUDA program to compute Matrix-Vector multiplication.

  Input       : None

  Output      : Execution time in seconds , Gflops achieved

  Created     : August-2013

  E-mail      : hpcfte@cdac.in

****************************************************************************/


#include<cstdio>
#include<hip/hip_runtime.h>
#include "serial_code.h"
#include "utils.h"
#include "cuda_utils.h"
#include "gpu_ops.h"
#include "host_ops.h"

#define SIZE 1024


int vlength, matRowSize , matColSize;
int size = SIZE;

/*main function*/
int main()
{
    // Vector length , Matrix Row and Col sizes..............
    vlength = matColSize = SIZE;
    matRowSize = SIZE;

    host_ops *host_system = new host_ops(matRowSize, matColSize, vlength);
    host_system->fill_with_random_data();

    int device_Count=get_DeviceCount();
    printf("\n\nNumber of Devices : %d\n\n", device_Count);



    gpu_ops *gpu_user = new gpu_ops(matRowSize, matColSize, vlength);
    gpu_user->set_device(0);
    gpu_user->start_event();
    gpu_user->allocate_memory();
    gpu_user->copy_to_device(host_system->host_Mat, host_system->host_Vect);
    gpu_user->launch_kernel();
    gpu_user->copy_to_host(host_system->host_ResVect);
    float time_sec = gpu_user->stop_event();



    // calling funtion for measuring Gflops & printing the result on screen
    print_on_screen("MAT VECT MULTIPLICATION",time_sec, calculate_gflops(time_sec, size),size,1);


    // CPU calculation..and checking error deviation....
    serial_code *cpu_user = new serial_code(matRowSize, matColSize, host_system->host_Mat, host_system->host_Vect, vlength, size);
    cpu_user->CPU_MatVectMult();
    relative_error(cpu_user->get_result(), host_system->host_ResVect, size);
    printf("\n ----------------------------------------------------------------------\n");

    host_system->_free();
    gpu_user->_free();
    cpu_user->_free();

    return 0;
}// end of main

