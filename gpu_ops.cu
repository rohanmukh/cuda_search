#include "hip/hip_runtime.h"
//
// Created by rm38 on 3/2/20.
//

#include "gpu_ops.h"
#include <cstdio>

#define BLOCKSIZE 16


/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication
//
/////////////////////////////////////////////////////////////////////////////////////////
__global__
void MatVectMultiplication(const double *_device_Mat, const double *_device_Vect, int _matRowSize, int _vlength, double *_device_ResVect)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;


    if(tindex < _matRowSize)
    {
        int i;int m= tindex * _vlength;
        _device_ResVect[tindex]=0.00;
        for(i=0; i < _vlength; i++)
            _device_ResVect[tindex]+= _device_Mat[m + i] * _device_Vect[i];
    }

    __syncthreads();

}//end of MatVect device function



/*function to launch kernel*/
void gpu_ops::launch_kernel() {
    // Launching kernel..........

    /* threads_per_block, blocks_per_grid  */
    int max=BLOCKSIZE*BLOCKSIZE;
    int BlocksPerGrid=matRowSize/max+1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(matRowSize%max==0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_Mat,device_Vect,matRowSize,vlength,device_ResVect);

}


void gpu_ops::get_device_property() {

    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    printf("Using device %d: %s \n", device, deviceProp.name);

}

void gpu_ops::set_device(int device_id, std::string message="") {
    // Device Selection, Device 1: Tesla C1060
    hipSetDevice(device_id);
}


void gpu_ops::allocate_memory() {
    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Mat, matRowSize*matColSize* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Vect, vlength* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_ResVect, matRowSize* sizeof(double)));
}


void gpu_ops::copy_to_device(double* host_Mat, double* host_Vect) {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Mat, (void*)host_Mat, matRowSize*matColSize*sizeof(double) ,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Vect, (void*)host_Vect,vlength*sizeof(double),hipMemcpyHostToDevice));
}



void gpu_ops::copy_to_host(double *host_ResVect) {
    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResVect,matRowSize*sizeof(double),hipMemcpyDeviceToHost));

}

void gpu_ops::_free() {
    /*free the memory from GPU */
    CUDA_SAFE_CALL(hipFree(device_Mat));
    CUDA_SAFE_CALL(hipFree(device_Vect));
    CUDA_SAFE_CALL(hipFree(device_ResVect));
    printf("mem freed\n");
}


void gpu_ops::start_event() {
    //event creation...
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventRecord (start, 0));
    CUDA_SAFE_CALL(hipEventCreate (&stop));
}

float gpu_ops::stop_event() {
    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));
    float Tsec= 1.0e-3*elapsedTime; // time in seconds
    return Tsec;
}

gpu_ops::gpu_ops(int device_id, int matRowSize, int matColSize, int vlength) {
    this->device_id = device_id;
    this->matRowSize = matRowSize;
    this->matColSize = matColSize;
    this->vlength = vlength;
    set_device(device_id);
    get_device_property();
}

