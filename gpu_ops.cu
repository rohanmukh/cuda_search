#include "hip/hip_runtime.h"
//
// Created by rm38 on 3/2/20.
//

#include "gpu_ops.h"
#include <cstdio>

#define BLOCKSIZE 16


/////////////////////////////////////////////////////////////////////////////////////////
//
// MatVect : this kernel will perform actual MatrixVector Multiplication
//
/////////////////////////////////////////////////////////////////////////////////////////
__global__
void MatVectMultiplication(const double *_device_Mat, const double *_device_Vect, int _matRowSize, int _vlength, double *_device_ResVect)
{
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int tidy = blockIdx.y*blockDim.y + threadIdx.y;
    int tindex=tidx+gridDim.x*BLOCKSIZE*tidy;


    if(tindex < _matRowSize)
    {
        int i;int m= tindex * _vlength;
        _device_ResVect[tindex]=0.00;
        for(i=0; i < _vlength; i++)
            _device_ResVect[tindex]+= _device_Mat[m + i] * _device_Vect[i];
    }

    __syncthreads();

}//end of MatVect device function



gpu_ops::gpu_ops(int matRowSize, int matColSize, double *host_Mat,
                         double *host_Vect, int vlength, int size) {
    this->matRowSize = matRowSize;
    this->matColSize = matColSize;
    this->host_Mat = host_Mat;
    this->host_Vect = host_Vect;
    this->vlength = vlength;
    this->size = size;
}


void gpu_ops::allocate_memory() {
    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Mat, matRowSize*matColSize* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Vect, vlength* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_ResVect, matRowSize* sizeof(double)));
}


void gpu_ops::copy_to_device() {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Mat, (void*)host_Mat, matRowSize*matColSize*sizeof(double) ,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Vect, (void*)host_Vect,vlength*sizeof(double),hipMemcpyHostToDevice));
}

void gpu_ops::launch_kernel(hipDeviceProp_t deviceProp) {
    // Launching kernell..........

    /* threads_per_block, blocks_per_grid  */
    int max=BLOCKSIZE*BLOCKSIZE;
    int BlocksPerGrid=matRowSize/max+1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(matRowSize%max==0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_Mat,device_Vect,matRowSize,vlength,device_ResVect);

}


void gpu_ops::get_data_to_host(double *host_ResVect) {
    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResVect,matRowSize*sizeof(double),hipMemcpyDeviceToHost));

}





void gpu_ops::_free() {
    /*free the memory from GPU */
    CUDA_SAFE_CALL(hipFree(device_Mat));
    CUDA_SAFE_CALL(hipFree(device_Vect));
    CUDA_SAFE_CALL(hipFree(device_ResVect));
    printf("mem freed\n");
}
