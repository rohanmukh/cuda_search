#include "hip/hip_runtime.h"
//
// Created by rm38 on 3/2/20.
//

#include "gpu_ops.h"
#include <cstdio>
#include <iostream>
#include "matrix_vector_op.cu"

/*function to launch kernel*/
void gpu_ops::launch_kernel() {
    // Launching kernel..........

    /* threads_per_block, blocks_per_grid  */
    int max=BLOCKSIZE*BLOCKSIZE;
    long BlocksPerGrid= batch_size / max + 1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(batch_size % max == 0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_database_A, device_database_B, device_database_probY,
            device_input_A, device_input_B, batch_size,dimension,device_ResDistance);

}


void gpu_ops::get_device_property() {
    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    printf("Using device %d: %s \n", device, deviceProp.name);

}

void gpu_ops::set_device(int _device_id, const std::string& message="") {
    // Device Selection, Device 1: Tesla C1060
    std::cout << "Setting Device :: " << _device_id << " for " << message << std::endl;
    hipSetDevice(_device_id);
}


void gpu_ops::allocate_memory() {
    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc((void**)&device_database_B, batch_size * dimension * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_database_A, batch_size * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_database_probY, batch_size * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_input_B, dimension * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_input_A, sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_ResDistance, batch_size * sizeof(double)));
}


void gpu_ops::copy_data_to_device(double* host_database_B, double* host_database_A, double* host_database_probY) {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_database_B, (void*)host_database_B, batch_size * dimension * sizeof(double) , hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_database_A, (void*)host_database_A, batch_size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_database_probY, (void*)host_database_probY, batch_size * sizeof(double), hipMemcpyHostToDevice));
}

void gpu_ops::copy_input_to_device(double* host_input_B, double* host_input_A) {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_input_B, (void*)host_input_B, dimension * sizeof(double) , hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_input_A, (void*)host_input_A, sizeof(double), hipMemcpyHostToDevice));
}

void gpu_ops::copy_result_to_host(double *host_ResVect) {
    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResDistance, batch_size * sizeof(double), hipMemcpyDeviceToHost));
}

void gpu_ops::_free() {
    /*free the memory from GPU */
    CUDA_SAFE_CALL(hipFree(device_database_B));
    CUDA_SAFE_CALL(hipFree(device_database_A));
    CUDA_SAFE_CALL(hipFree(device_input_B));
    CUDA_SAFE_CALL(hipFree(device_input_A));
    CUDA_SAFE_CALL(hipFree(device_ResDistance));
}


void gpu_ops::start_event() {
    //event creation...
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventRecord (start, 0));
    CUDA_SAFE_CALL(hipEventCreate (&stop));
}

float gpu_ops::stop_event() {
    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));
    float Tsec= 1.0e-3*elapsedTime; // time in seconds
    return Tsec;
}

gpu_ops::gpu_ops(int device_id, long batch_size, int dimension) {
    this->device_id = device_id;
    this->batch_size = batch_size;
    this->dimension = dimension;
    set_device(device_id, "Initialization");
    get_device_property();
    allocate_memory();
}

