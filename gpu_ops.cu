#include "hip/hip_runtime.h"
//
// Created by rm38 on 3/2/20.
//

#include "gpu_ops.h"
#include <cstdio>
#include "matrix_vector_op.cu"

/*function to launch kernel*/
void gpu_ops::launch_kernel() {
    // Launching kernel..........

    /* threads_per_block, blocks_per_grid  */
    int max=BLOCKSIZE*BLOCKSIZE;
    int BlocksPerGrid=matRowSize/max+1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(matRowSize%max==0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_Mat,device_Vect,matRowSize,vlength,device_ResVect);

}


void gpu_ops::get_device_property() {

    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    printf("Using device %d: %s \n", device, deviceProp.name);

}

void gpu_ops::set_device(int device_id, std::string message="") {
    // Device Selection, Device 1: Tesla C1060
    hipSetDevice(device_id);
}


void gpu_ops::allocate_memory() {
    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Mat, matRowSize*matColSize* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_Vect, vlength* sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc( (void**)&device_ResVect, matRowSize* sizeof(double)));
}


void gpu_ops::copy_to_device(double* host_Mat, double* host_Vect) {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Mat, (void*)host_Mat, matRowSize*matColSize*sizeof(double) ,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_Vect, (void*)host_Vect,vlength*sizeof(double),hipMemcpyHostToDevice));
}



void gpu_ops::copy_to_host(double *host_ResVect) {
    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_ResVect,matRowSize*sizeof(double),hipMemcpyDeviceToHost));

}

void gpu_ops::_free() {
    /*free the memory from GPU */
    CUDA_SAFE_CALL(hipFree(device_Mat));
    CUDA_SAFE_CALL(hipFree(device_Vect));
    CUDA_SAFE_CALL(hipFree(device_ResVect));
    printf("mem freed\n");
}


void gpu_ops::start_event() {
    //event creation...
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventRecord (start, 0));
    CUDA_SAFE_CALL(hipEventCreate (&stop));
}

float gpu_ops::stop_event() {
    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));
    float Tsec= 1.0e-3*elapsedTime; // time in seconds
    return Tsec;
}

gpu_ops::gpu_ops(int device_id, int matRowSize, int matColSize, int vlength) {
    this->device_id = device_id;
    this->matRowSize = matRowSize;
    this->matColSize = matColSize;
    this->vlength = vlength;
    set_device(device_id);
    get_device_property();
}

