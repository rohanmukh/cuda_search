#include "hip/hip_runtime.h"
//
// Created by rm38 on 3/2/20.
//

#include "single_gpu_manager.h"
#include <cstdio>
#include <iostream>
#include "matrix_vector_op.cu"

/*function to launch kernel*/
void single_gpu_manager::launch_kernel() {
    // Launching kernel..........

    /* threads_per_block, blocks_per_grid  */
    int max=BLOCKSIZE*BLOCKSIZE;
    long BlocksPerGrid= device_data_size / max + 1;
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    if(device_data_size % max == 0)BlocksPerGrid--;
    dim3 dimGrid(1,BlocksPerGrid);
    check_block_grid_dim(deviceProp,dimBlock,dimGrid);

    MatVectMultiplication<<<dimGrid,dimBlock>>>(device_database_A, device_database_B, device_database_probY,
            device_input_A, device_input_B, device_data_size,dimension,device_result_vector);

}


void single_gpu_manager::get_device_property() {
    int device;
    // Current Device Detection
    hipGetDevice(&device);
    hipGetDeviceProperties(&deviceProp,device);
    // printf("Using device %d: %s \n", device, deviceProp.name);

}

void single_gpu_manager::set_device(int _device_id, const std::string& message="") {
    // Device Selection, Device 1: Tesla C1060
    // std::cout << "Setting Device :: " << _device_id << " for " << message << std::endl;
    hipSetDevice(_device_id);
}


void single_gpu_manager::allocate_memory() {
    //allocating memory on GPU
    CUDA_SAFE_CALL(hipMalloc((void**)&device_database_B, device_data_size * dimension * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_database_A, device_data_size * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_database_probY, device_data_size * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_input_B, dimension * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_input_A, sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&device_result_vector, device_data_size * sizeof(double)));
}


void single_gpu_manager::copy_data_to_device(long offset, float* host_database_B, float* host_database_A, float* host_database_probY) {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)(device_database_B + offset*dimension), (void*)host_database_B, batch_size * dimension * sizeof(double) , hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)(device_database_A + offset), (void*)host_database_A, batch_size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)(device_database_probY + offset), (void*)host_database_probY, batch_size * sizeof(double), hipMemcpyHostToDevice));
}

void single_gpu_manager::copy_input_to_device(float* host_input_B, float* host_input_A) {
    //moving data from CPU to GPU
    CUDA_SAFE_CALL(hipMemcpy((void*)device_input_B, (void*)host_input_B, dimension * sizeof(double) , hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy((void*)device_input_A, (void*)host_input_A, sizeof(double), hipMemcpyHostToDevice));
}

void single_gpu_manager::copy_result_to_host(float *host_ResVect) {
    //retriving result from device
    CUDA_SAFE_CALL(hipMemcpy((void*)host_ResVect, (void*)device_result_vector, device_data_size * sizeof(double), hipMemcpyDeviceToHost));
}

void single_gpu_manager::_free() {
    /*free the memory from GPU */
    CUDA_SAFE_CALL(hipFree(device_database_B));
    CUDA_SAFE_CALL(hipFree(device_database_A));
    CUDA_SAFE_CALL(hipFree(device_database_probY));
    CUDA_SAFE_CALL(hipFree(device_input_B));
    CUDA_SAFE_CALL(hipFree(device_input_A));
    CUDA_SAFE_CALL(hipFree(device_result_vector));
}


void single_gpu_manager::start_event() {
    //event creation...
    CUDA_SAFE_CALL(hipEventCreate (&start));
    CUDA_SAFE_CALL(hipEventRecord (start, 0));
    CUDA_SAFE_CALL(hipEventCreate (&stop));
}

double single_gpu_manager::stop_event() {
    CUDA_SAFE_CALL(hipEventRecord (stop, 0));
    CUDA_SAFE_CALL(hipEventSynchronize (stop));
    CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));
    double Tsec= 1.0e-3*elapsedTime; // time in seconds
    return Tsec;
}

single_gpu_manager::single_gpu_manager(int device_id, long device_num_batches, long batch_size, int dimension) {
    this->device_id = device_id;

    this->device_num_batches = device_num_batches;
    this->batch_size = batch_size;
    this->device_data_size = device_num_batches*batch_size;

    this->dimension = dimension;
    set_device(device_id, "Initialization");
    get_device_property();
    allocate_memory();
}
